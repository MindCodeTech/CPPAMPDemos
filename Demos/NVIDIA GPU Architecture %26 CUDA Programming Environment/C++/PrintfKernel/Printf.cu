#include "hip/hip_runtime.h"
/*
 * Example demonstrates how to use printf command in CUDA kernel
 */

#include <hip/hip_runtime.h>
#include <>

#include <assert.h>
#include <iostream>
#include <stdio.h>
#include <string>

using namespace std;

// printf is only supported for compute capability 2.0 and higher
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
#define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif

// Forward declarations
hipError_t printfTest();
void		SetDevice();

// Print kernel
__global__ void printfKernel(float f)
{
	float data = f * threadIdx.x;
	printf("Thread %d, f = %f.\n", threadIdx.x, data);
}

int main()
{
	// Show device info and pick the best available device
	SetDevice();

	// Call kernel
	hipError_t ce = printfTest();
	if (ce != hipSuccess)
	{
		cerr << "printfTest failed!" << endl;
		return 1;
	}

	// Reset device before exiting for profiling and tracing tools to show compete traces
	ce = hipDeviceReset();
	if (ce != hipSuccess)
	{
		cerr << "hipDeviceReset failed!" << endl;
		return 1;
	}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t printfTest()
{
	hipError_t ce = hipSuccess;

	printfKernel<<< 1, 64 >>>(1.2345f);
	ce = hipDeviceSynchronize();

    if (ce != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << to_string(ce) << " after launching printfKernel!\n" << endl;
        goto Error;
    }

Error:
	return ce;
}

// Show device information
/*
Using device: 0
	Name:                    Quadro 5000M
	Compute version:         2.0
	Global memory:           2047.69 mb
	Const memory:            64 kb
	L2 cache size:           512 kb
	Clock rate:              810 mhz
	Timeout enabled:         true
	Multiprocessors:         10
	Max grid size:           65535 : 65535 : 65535
	Max threads per SM:      1536
	Max threads per block:   1024
	Registers per block:     32768
	Shared memory per block: 48 kb
	Memory bus width:        256 bits
	Memory clock rate:       1200 mhz
	Compute mode:            Default
*/
void SetDevice()
{
	// Only show stats on the first run of this code
	static bool showStats = true;

	if (!showStats)
		return;
	
	showStats = false;

	// Set device and display properties
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		cerr << "ERROR: Your system does not have CUDA." << endl;
		exit(EXIT_FAILURE);
	}

	int device;
	for (device = 0; device < deviceCount; ++device)
	{
		hipDeviceProp_t p;
		hipError_t ce = hipGetDeviceProperties(&p, device);

		if (ce != hipSuccess)
		{
			cerr << "ERROR: Device query failed." << endl;
			exit(EXIT_FAILURE);
		}

		cout << "\n\nUsing device: " << device << endl;
		cout << "   Name:                    " << p.name << endl;
		cout << "   Compute capability:      " << p.major << "." << p.minor << endl;
		cout << "   Warp Size:               " << p.warpSize << endl;
		cout << "   Global memory:           " << p.totalGlobalMem / static_cast<float>(1024 * 1024) << " mb" << endl;
		cout << "   Const memory:            " << p.totalConstMem / static_cast<float>(1024) << " kb" << endl;
		cout << "   L2 cache size:           " << p.l2CacheSize / static_cast<float>(1024) << " kb" << endl;
		cout << "   Clock rate:              " << p.clockRate / 1000.f << " mhz"<< endl;
		cout << "   Timeout enabled:         " << (p.kernelExecTimeoutEnabled == 1 ? "true" : "false") << endl;
		cout << "   Multiprocessors:         " << p.multiProcessorCount << endl;
		cout << "   Max grid size:           " << p.maxGridSize[0] << " : " << p.maxGridSize[1] << " : " << p.maxGridSize[2] << endl;
		cout << "   Max threads per SM:      " << p.maxThreadsPerMultiProcessor << endl;
		cout << "   Max threads per block:   " << p.maxThreadsPerBlock << endl;
		cout << "   Registers per block:     " << p.regsPerBlock << endl;
		cout << "   Shared memory per block: " << p.sharedMemPerBlock / static_cast<float>(1024) << " kb" << endl;
		cout << "   Memory bus width:        " << p.memoryBusWidth << " bits" << endl;
		cout << "   Memory clock rate:       " << p.memoryClockRate / 1000.f << " mhz" << endl;
		cout << "   Compute mode:            " << (p.computeMode == 0 ? "Default" : "Exclusive or Prohibitive") << endl;
		cout << "   PCI Bus id:              " << p.pciBusID << endl;
		cout << "   PCI Device id:           " << p.pciDeviceID << endl;
		cout << "   PCI Domain id:           " << p.pciDomainID << endl;
		cout << "   Number of async engines: " << p.asyncEngineCount << endl;
		cout << "   Discrete GPU:            " << (p.integrated == 0 ? "Yes" : "No") << endl;
		cout << "   Can map host memory:     " << (p.canMapHostMemory == 1 ? "Yes" : "No") << endl;
		cout << "   Concurrent kernels:      " << (p.concurrentKernels == 1 ? "Yes" : "No") << endl;
		cout << "   ECC supported:           " << (p.ECCEnabled == 1 ? "Yes" : "No") << endl;
		cout << "   Unified addressing:      " << (p.unifiedAddressing == 1 ? "Yes" : "No") << endl;
		cout << endl << endl;

		// Set best device here
		if (device == 0)
		{			
			ce = hipSetDevice(0);
			if (ce != hipSuccess)
			{
				cerr << "ERROR: failed to set CUDA device!" << endl;
				exit(EXIT_FAILURE);
			}
		}
	}
}
